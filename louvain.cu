#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <sstream>
#include <unordered_map>
#include <chrono>

#define THREADS_PER_BLOCK 256

__global__ void computeModularityGain(int *row_ptr, int *col_idx, int *community, int numNodes) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= numNodes) return;

    int bestCommunity = community[tid];
    float bestGain = 0.0f;

    for (int i = row_ptr[tid]; i < row_ptr[tid + 1]; i++) {
        int neighbor = col_idx[i];
        int newCommunity = community[neighbor];

        float gain = 0.1f * (newCommunity != bestCommunity);
        if (gain > bestGain) {
            bestGain = gain;
            bestCommunity = newCommunity;
        }
    }
    community[tid] = bestCommunity;
}

__global__ void aggregate_communities(int *row_ptr, int *col_idx, int *d_comm, float *d_weights, int numNodes) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= numNodes) return;

    int comm = d_comm[tid];

    for (int i = row_ptr[tid]; i < row_ptr[tid + 1]; i++) {
        int neighbor = col_idx[i];
        int neighborComm = d_comm[neighbor];

        if (comm == neighborComm) {
            atomicAdd(&d_weights[comm], 1.0f);
        }
    }
}


void loadGraphCSR(const std::string &filename, std::vector<int> &row_ptr, std::vector<int> &col_idx, int &numNodes) {
    std::ifstream file(filename);
    std::unordered_map<int, std::vector<int>> adjList;
    int maxNode = 0;

    std::string line;
    while (std::getline(file, line)) {
        if (line[0] == '#') continue;
        std::istringstream iss(line);
        int u, v;
        if (!(iss >> u >> v)) continue;

        adjList[u].push_back(v);
        adjList[v].push_back(u);
        maxNode = std::max(maxNode, std::max(u, v));
    }
    file.close();

    numNodes = maxNode + 1;
    row_ptr.resize(numNodes + 1, 0);

    for (int i = 0; i < numNodes; i++) {
        row_ptr[i + 1] = row_ptr[i] + adjList[i].size();
    }
    for (int i = 0; i < numNodes; i++) {
        col_idx.insert(col_idx.end(), adjList[i].begin(), adjList[i].end());
    }
}

void louvainCUDA(std::vector<int> &row_ptr, std::vector<int> &col_idx, std::vector<int> &communities, int numNodes) {
    int *d_row_ptr, *d_col_idx, *d_community;

    hipMalloc(&d_row_ptr, (numNodes + 1) * sizeof(int));
    hipMalloc(&d_col_idx, col_idx.size() * sizeof(int));
    hipMalloc(&d_community, numNodes * sizeof(int));

    hipMemcpy(d_row_ptr, row_ptr.data(), (numNodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_idx, col_idx.data(), col_idx.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_community, communities.data(), numNodes * sizeof(int), hipMemcpyHostToDevice);

    int blocks = (numNodes + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    computeModularityGain<<<blocks, THREADS_PER_BLOCK>>>(d_row_ptr, d_col_idx, d_community, numNodes);
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(communities.data(), d_community, numNodes * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_row_ptr);
    hipFree(d_col_idx);
    hipFree(d_community);

    std::cout << "CUDA Louvain execution time: " << milliseconds << " ms\n";
}

int main() {
    std::string filename = "soc-LiveJournal1.txt";
    std::vector<int> row_ptr, col_idx;
    int numNodes;

    auto start = std::chrono::high_resolution_clock::now();
    loadGraphCSR(filename, row_ptr, col_idx, numNodes);
    auto stop = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> load_time = stop - start;
    std::cout << "Graph loading time: " << load_time.count() << " seconds\n";

    std::vector<int> communities(numNodes);
    for (int i = 0; i < numNodes; i++) {
        communities[i] = i;
    }

    louvainCUDA(row_ptr, col_idx, communities, numNodes);

    float *d_weights;
    int *d_newGraph, *d_comm, *d_adj;

    hipMalloc(&d_weights, numNodes * sizeof(float));
    hipMemset(d_weights, 0, numNodes * sizeof(float));
    hipMalloc(&d_newGraph, numNodes * numNodes * sizeof(int));
    hipMalloc(&d_comm, numNodes * sizeof(int));
    hipMalloc(&d_adj, col_idx.size() * sizeof(int));

    hipMemcpy(d_comm, communities.data(), numNodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_adj, col_idx.data(), col_idx.size() * sizeof(int), hipMemcpyHostToDevice);

    int blocks = (numNodes + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    
    int *d_row_ptr, *d_col_idx;
    hipMalloc(&d_row_ptr, (numNodes + 1) * sizeof(int));
    hipMalloc(&d_col_idx, col_idx.size() * sizeof(int));
    hipMemcpy(d_row_ptr, row_ptr.data(), (numNodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_idx, col_idx.data(), col_idx.size() * sizeof(int), hipMemcpyHostToDevice);

    aggregate_communities<<<blocks, THREADS_PER_BLOCK>>>(d_row_ptr, d_col_idx, d_comm, d_weights, numNodes);
    hipDeviceSynchronize();

    hipMemcpy(communities.data(), d_comm, numNodes * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Updated communities (first 10 nodes):\n";
    for (int i = 0; i < 10; i++) {
        std::cout << "Node " << i << " -> Community " << communities[i] << "\n";
    }

    std::cout << "Phase 2: Printing community weights...\n";

    std::vector<float> h_weights_out(numNodes);

    hipDeviceSynchronize();
    hipMemcpy(h_weights_out.data(), d_weights, numNodes * sizeof(float), hipMemcpyDeviceToHost);
    
    std::cout << "Community weights (first 100):\n";
    for (int i = 0; i < 100; i++) {
        std::cout << "Community " << i << " has total weight: " << h_weights_out[i] << "\n";
    }

    hipFree(d_weights);
    hipFree(d_newGraph);
    hipFree(d_comm);
    hipFree(d_adj);
    
    return 0;
}